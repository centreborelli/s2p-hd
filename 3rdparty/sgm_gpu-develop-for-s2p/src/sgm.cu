/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <stdlib.h>
#include <iostream>
#include <chrono>

#include "sgm.hpp"
#include "census_transform.hpp"
#include "path_aggregation.hpp"
#include "winner_takes_all.hpp"

namespace sgm {

template <typename T, size_t MAX_DISPARITY>
class SemiGlobalMatching<T, MAX_DISPARITY>::Impl {

private:
	CensusTransform<T> m_census_left;
	CensusTransform<T> m_census_right;
	PathAggregation<MAX_DISPARITY> m_path_aggregation;
	WinnerTakesAll<MAX_DISPARITY> m_winner_takes_all;

public:
	Impl()
		: m_census_left()
		, m_census_right()
		, m_path_aggregation()
		, m_winner_takes_all()
	{ }

	void enqueue(
		output_type *dest_left,
		output_type *dest_right,
		const input_type *src_left,
		const input_type *src_right,
		int width,
		int height,
		int src_pitch,
		int dst_pitch,
		const StereoSGM::Parameters& param,
		hipStream_t stream)
	{

        std::chrono::steady_clock::time_point begin_census_transform_proc = std::chrono::steady_clock::now();
		m_census_left.enqueue(
			src_left, width, height, src_pitch, stream);
        std::chrono::steady_clock::time_point end_census_transform_left = std::chrono::steady_clock::now();
        std::chrono::steady_clock::time_point begin_census_transform_right = std::chrono::steady_clock::now();
		m_census_right.enqueue(
			src_right, width, height, src_pitch, stream);
        std::chrono::steady_clock::time_point end_census_transform_right = std::chrono::steady_clock::now();
	if (param.verbose)
		std::cout << "\t\tTime spent census transf left = " << std::chrono::duration_cast<std::chrono::microseconds>(end_census_transform_left - begin_census_transform_proc).count() << "[us]" << std::endl;
	if (param.verbose)
		std::cout << "\t\tTime spent census transf right = " << std::chrono::duration_cast<std::chrono::microseconds>(end_census_transform_right - begin_census_transform_right).count() << "[us]" << std::endl;

        std::chrono::steady_clock::time_point begin_m_path_aggregation = std::chrono::steady_clock::now();
        m_path_aggregation.enqueue(
			m_census_left.get_output(),
			m_census_right.get_output(),
			width, height,
			param.path_type, param.P1, param.P2, param.min_disp,
			stream);
        std::chrono::steady_clock::time_point end_m_path_aggregation = std::chrono::steady_clock::now();
	if (param.verbose)
		std::cout << "\t\tTime spent path aggregation = " << std::chrono::duration_cast<std::chrono::microseconds>(end_m_path_aggregation - begin_m_path_aggregation).count() << "[us]" << std::endl;

        std::chrono::steady_clock::time_point begin_m_winner_takes_all = std::chrono::steady_clock::now();
        m_winner_takes_all.enqueue(
			dest_left, dest_right,
			m_path_aggregation.get_output(),
			width, height, dst_pitch,
			param.uniqueness, param.subpixel, param.path_type,
			stream);
        std::chrono::steady_clock::time_point end_m_winner_takes_all = std::chrono::steady_clock::now();
	if (param.verbose)
		std::cout << "\t\tTime spent winner takes all = " << std::chrono::duration_cast<std::chrono::microseconds>(end_m_winner_takes_all - begin_m_winner_takes_all).count() << "[us]" << std::endl;

    }

};


template <typename T, size_t MAX_DISPARITY>
SemiGlobalMatching<T, MAX_DISPARITY>::SemiGlobalMatching()
	: m_impl(new Impl())
{ }

template <typename T, size_t MAX_DISPARITY>
SemiGlobalMatching<T, MAX_DISPARITY>::~SemiGlobalMatching() = default;


template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::execute(
	output_type *dest_left,
	output_type *dest_right,
	const input_type *src_left,
	const input_type *src_right,
	int width,
	int height,
	int src_pitch,
	int dst_pitch,
	const StereoSGM::Parameters& param)
{
	m_impl->enqueue(
		dest_left, dest_right,
		src_left, src_right,
		width, height,
		src_pitch, dst_pitch,
		param,
		0);
	hipStreamSynchronize(0);
}

template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::enqueue(
	output_type *dest_left,
	output_type *dest_right,
	const input_type *src_left,
	const input_type *src_right,
	int width,
	int height,
	int src_pitch,
	int dst_pitch,
	const StereoSGM::Parameters& param,
	hipStream_t stream)
{
	m_impl->enqueue(
		dest_left, dest_right,
		src_left, src_right,
		width, height,
		src_pitch, dst_pitch,
		param,
		stream);
}


template class SemiGlobalMatching<uint8_t,   64>;
template class SemiGlobalMatching<uint8_t,  128>;
template class SemiGlobalMatching<uint8_t,  256>;
template class SemiGlobalMatching<uint8_t,  512>;
template class SemiGlobalMatching<uint16_t,  64>;
template class SemiGlobalMatching<uint16_t, 128>;
template class SemiGlobalMatching<uint16_t, 256>;
template class SemiGlobalMatching<uint16_t, 512>;

}
